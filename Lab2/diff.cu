
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define BLOCKSIZE 4 // Number of threads in each thread block
 
// CUDA kernel. Each thread takes care of one element of a 
__global__ void diffKernel( float *in, float *out, int n )
{
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    // Write the kernel to implement the diff operation on an array
    if(threadId < n - 1)
        out[threadId] = in[threadId + 1] - in[threadId];
}  
 
int main( int argc, char* argv[] )
{
    hipDeviceReset();

    // Size of vectors
    int i;
    float input[] = {4, 5, 6, 7, 19, 10, 0, 4, 2, 3, 1, 7, 9, 11, 45, 23, 99, 29};
    int n = sizeof(input) / sizeof(float); //careful, this usage only works with statically allocated arrays, NOT dynamic arrays

    // Host input vectors
    float *h_in = input;
    //Host output vector
    float *h_out = (float *) malloc((n - 1) * sizeof(float));
 
    // Device input vectors
    float *d_in;
    //Device output vector
    float *d_out;
 
    // Size, in bytes, of each vector
    size_t bytes = n * sizeof(float);
 
    // Allocate memory for each vector on GPU
    hipMalloc(&d_in, bytes);
    hipMalloc(&d_out, bytes - sizeof(float));
 
    // Copy host data to device
    hipMemcpy( d_in, h_in, bytes, hipMemcpyHostToDevice);

    // TODO: setup the blocksize and gridsize and launch the kernel below.
    int blocksize = BLOCKSIZE;
    // Number of threads in each thread block

    // Number of thread blocks in grid
    int gridsize = (int) ceil((double)n / (double)blocksize);

    // Execute the kernel
    diffKernel<<<gridsize, blocksize>>>(d_in, d_out, n);
 
    // Copy array back to host
    hipMemcpy( h_out, d_out, bytes - sizeof(float), hipMemcpyDeviceToHost );
 
    // Show the result
    printf("The original array is: ");
    for(i = 0; i < n; i ++)
        printf("%4.0f,", h_in[i] );    
    
    printf("\n\nThe diff     array is: ");
    for(i = 0; i < n - 1; i++)
        printf("%4.0f,", h_out[i] );    
    puts("");
    
    // Release device memory
    hipFree(d_in);
    hipFree(d_out);
 
    // Release host memory
    free(h_out);
 
    return 0;
}
