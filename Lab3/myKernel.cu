#include "hip/hip_runtime.h"
#include "myKernel.h"

__global__ void kernel( int *a, int dimx, int dimy )
{
    int ix   = blockIdx.x*blockDim.x + threadIdx.x;
    int iy   = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = iy*dimx + ix;

    a[idx]  = a[idx]+1;
}

// Please implement the following kernels2 through kernel6,
// in order to meet the requirements in the write-ups. 
__global__ void kernel2( int *a, int dimx, int dimy )
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    // make sure we are in the array
    if(col >= dimx || row >= dimy)
        return;

    int outIndex = row * dimx + col;
    a[outIndex] = blockIdx.y * gridDim.x + blockIdx.x;
}

__global__ void kernel3( int *a, int dimx, int dimy )
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    // make sure we are in the array
    if(col >= dimx || row >= dimy)
        return;

    int outIndex = row * dimx + col;
    a[outIndex] = outIndex;
}

__global__ void kernel4( int *a, int dimx, int dimy )
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    // make sure we are in the array
    if(col >= dimx || row >= dimy)
        return;

    int outIndex = row * dimx + col;
    a[outIndex] = threadIdx.y * blockDim.x + threadIdx.x;
}

__global__ void kernel5( int *a, int dimx, int dimy )
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    // make sure we are in the array
    if(col >= dimx || row >= dimy)
        return;

    int outIndex = row * dimx + col;
    a[outIndex] = blockIdx.y;
}

__global__ void kernel6( int *a, int dimx, int dimy )
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    // make sure we are in the array
    if(col >= dimx || row >= dimy)
        return;

    int outIndex = row * dimx + col;
    a[outIndex] = blockIdx.x;
}


