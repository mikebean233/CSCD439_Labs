
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#define R 4
#define C 40 

/*
 * It returns the length of a string pointed by pointer s,
 * It acts like the cpu strlen() function
 */
__device__ int gpu_strlen(char * s)
{
    int len = -1;
    while(s[++len] != '\0')
    {}

    return len;
}

/*
 * It returns 0 if input character ch is NOT an alphabetical letter
 * Otherwise, it returns one.
 */
__device__ int gpu_isAlpha(char ch)
{
    char* upperCase = "abcdefghijklmnopqrstuvwxyz";
    char* lowerCase = "ABCDEFGHIJKLMNOPQRSTUVWXUZ";
    int i = 0;
    
    for(i = 0; i < 26; ++i){
        if(upperCase[i] == ch || lowerCase[i] == ch)
            return 1;
    }
    return 0;
}

/* Cuda kernel to count number of words in each line of text pointed by a.
 * The output is stored back in 'out' array.
 * numLine specifies the num of lines in a, maxLineLen specifies the maximal
 * num of characters in one line of text.
 */
__global__ void wordCount( char **a, int **out, int numLine, int maxLineLen )
{
    unsigned int row = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned int col = blockDim.x * blockIdx.x + threadIdx.x;

    if(col > maxLineLen - 1 || row > numLine - 1 || col == 0)
        return;
    
    out[row][col] = (!gpu_isAlpha(a[row][col]) && gpu_isAlpha(a[row][col - 1])) ? 1 : 0;
}  

/* Print out the all lines of text in a on stdout
 */ 
void printArr( char **a, int lines )
{
    int i;
    for(i=0; i<lines; i++)
    {
        printf("%s\n", a[i]);
    }
}


int main()
{
    int i; 
    char **d_in, **h_in, **h_out;
    int h_count_in[R][C], **h_count_out, **d_count_in;

    //allocate
    h_in = (char **)malloc(R * sizeof(char *));
    h_out = (char **)malloc(R * sizeof(char *));
    h_count_out = (int **)malloc(R * sizeof(int *));

    hipMalloc((void ***)&d_in, sizeof(char *) * R);
    hipMalloc((void ***)&d_count_in, sizeof(int *) * R);

    //alocate for string data
    for(i = 0; i < R; ++i) 
    {
        hipMalloc((void **) &h_out[i],C * sizeof(char));
        h_in[i]=(char *)calloc(C, sizeof(char));//allocate or connect the input data to it
        strcpy(h_in[i], "good morning and I'm a good student!");
        hipMemcpy(h_out[i], h_in[i], strlen(h_in[i]) + 1, hipMemcpyHostToDevice);
    }
    hipMemcpy(d_in, h_out, sizeof(char *) * R,hipMemcpyHostToDevice);

    //alocate for output occurrence
    for(i = 0; i < R; ++i) 
    {
        hipMalloc((void **) &h_count_out[i], C * sizeof(int));
        hipMemset(h_count_out[i], 0, C * sizeof(int));
    }
    hipMemcpy(d_count_in, h_count_out, sizeof(int *) * R,hipMemcpyHostToDevice);

    printArr(h_in, R);
    printf("\n\n");
     
    //set up kernel configuartion variables
    dim3 grid, block;
    block.x = 2;
    block.y = 2;
    grid.x  = ceil((float)C / block.x);
    grid.y  = ceil((float)R / block.y); //careful must be type cast into float, otherwise, integer division used
    //printf("grid.x = %d, grid.y=%d\n", grid.x, grid.y );

    //launch kernel
    wordCount<<<grid, block>>>( d_in, d_count_in, R, C);

    //copy data back from device to host
    for(i = 0; i < R; ++i) {
        hipMemcpy(h_count_in[i], h_count_out[i], sizeof(int) * C,hipMemcpyDeviceToHost);
    }
    printf("Occurrence array obtained from device:\n");

    for(i = 0; i < R; i ++) {
        for(int j = 0; j < C; j ++)
            printf("%4d", h_count_in[i][j]);
        printf("\n");
    }
 
    return 0;
}

